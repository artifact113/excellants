#include "hip/hip_runtime.h"
/****************************************
 * Colony.cu                            *
 * Peter Ahrens                         *
 * Main ACO procedures                  *
 ****************************************/

#include "Colony.h"

//Constructor: Sets defaults and allocates memory.
Colony::Colony(thrust::host_vector<float> newDistances, int newNumCities, int newNumAnts)
{
  //defaults
  beta = 2;
  rho = 0.1;
  //world vars
  reps = 0;
  distances.assign(newDistances.begin(),newDistances.end());
  numCities = newNumCities;
  probabilities = thrust::device_vector<float>(numCities*numCities);
  pheromones = thrust::device_vector<float>(numCities*numCities);
  //ant vars
  numAnts = newNumAnts;
  antDistances = thrust::device_vector<float>(numAnts);
  antVisits = thrust::device_vector<float>(numCities*numAnts);
  toVisit = thrust::device_vector<int>(numCities*numAnts);
  antTours = thrust::device_vector<int>(numCities*numAnts);
  iterBestDist = std::numeric_limits<float>::max() - 1;
  globBestDist = std::numeric_limits<float>::max();
  iterBestTour = thrust::device_vector<int>(numCities);
  globBestTour = thrust::device_vector<int>(numCities);
  //maps and keys
  ACMapF = thrust::device_vector<int>(numAnts);
  ACMapL = thrust::device_vector<int>(numAnts);
  tourMap = thrust::device_vector<int>(numAnts);
  distMap = thrust::device_vector<int>(numCities*numAnts);
  ACKey = thrust::device_vector<int>(numAnts*numCities);
  ARepeatCMap = thrust::device_vector<int>(numAnts*numCities);
  CCKey = thrust::device_vector<int>(numCities*numCities);
  //scratch variables
  AFloat = thrust::device_vector<float>(numAnts);
  CInt = thrust::device_vector<float>(numCities);
  AInt = thrust::device_vector<int>(numAnts);
  ACInt = thrust::device_vector<int>(numAnts*numCities);
  ACInt2 = thrust::device_vector<int>(numAnts*numCities);
  ACInt3 = thrust::device_vector<int>(numAnts*numCities);
  ACFloat = thrust::device_vector<float>(numAnts*numCities);
  CCFloat = thrust::device_vector<float>(numCities*numCities);
  AUnsignedInt = thrust::device_vector<unsigned int>(numAnts);
  //Random numbers
  ARandom = thrust::device_vector<unsigned int>(numAnts);
  ACRandom = thrust::device_vector<unsigned int>(numAnts*numCities);
}

//initialize: Initializes data, creates maps and keys, performs standard ACO initialization steps etc. 
void Colony::initialize()
{
  //seed the random numbers
  thrust::transform(thrust::make_counting_iterator(0),
		    thrust::make_counting_iterator(numAnts*numCities),
		    thrust::make_counting_iterator(time(NULL)),ACRandom.begin(),thrust::multiplies<int>());
  thrust::transform(thrust::make_counting_iterator(0),
		    thrust::make_counting_iterator(numAnts),
		    thrust::make_counting_iterator(time(NULL)),
		    ARandom.begin(),
		    thrust::multiplies<int>());
  //constant seeds
  //thrust::transform(thrust::make_counting_iterator(0),thrust::make_counting_iterator(numAnts*numCities),thrust::make_constant_iterator(1),ACRandom.begin(),thrust::multiplies<int>());
  //thrust::transform(thrust::make_counting_iterator(0),thrust::make_counting_iterator(numAnts),thrust::make_constant_iterator(1),ARandom.begin(),thrust::multiplies<int>());
  thrust::transform(ACRandom.begin(),
		    ACRandom.end(),
		    ACRandom.begin(),
		    randStep());
  thrust::transform(ARandom.begin(),
		    ARandom.end(),
		    ARandom.begin(),
		    randStep());
  //create maps and keys
  //CCKey
  thrust::sequence(ACInt.begin(),
		   ACInt.begin() + numCities, 
		   0,
		   numCities);
  thrust::scatter(thrust::make_constant_iterator(1,0),
		  thrust::make_constant_iterator(1,numCities),
		  ACInt.begin(),
		  CCKey.begin());
  thrust::inclusive_scan(CCKey.begin(),
			 CCKey.end(),
			 CCKey.begin());
  //ACMapF
  thrust::sequence(ACMapF.begin(),
		   ACMapF.end(),
		   0,
		   numCities);
  //ACMapL
  thrust::transform(ACMapF.begin(),
		    ACMapF.end(),
		    thrust::make_constant_iterator(numCities-1),
		    ACMapL.begin(), 
		    thrust::plus<int>());
  //ACKey
  thrust::scatter(thrust::make_constant_iterator(1,0),
		  thrust::make_constant_iterator(1,numAnts),
		  ACMapF.begin(),
		  ACKey.begin());
  thrust::inclusive_scan(ACKey.begin(),
			 ACKey.end(),
			 ACKey.begin());
  thrust::transform(ACKey.begin(), 
		    ACKey.end(), 
		    thrust::make_constant_iterator(-1), 
		    ACKey.begin(),
		    thrust::plus<int>());
  //distMap
  thrust::fill(distMap.begin(),
	       distMap.end(),
	       1);
  thrust::inclusive_scan_by_key(ACKey.begin(),
				ACKey.end(),
				distMap.begin(), 
				distMap.begin());
  thrust::scatter(thrust::make_constant_iterator(0,0),
		  thrust::make_constant_iterator(0,numAnts),
		  ACMapL.begin(),
		  distMap.begin());
  thrust::transform(ACKey.begin(),
		    ACKey.end(),
		    distMap.begin(),
		    distMap.begin(),
		    saxpy_functor(numCities));
  //ARepeatCMap
  thrust::exclusive_scan_by_key(ACKey.begin(),
				ACKey.end(),
				thrust::make_constant_iterator(1),
				ARepeatCMap.begin());
  //ACO Initialize
  computeParameters();
  thrust::fill(pheromones.begin(),
	       pheromones.end(),
	       initialPheromone);
  computeProbabilities();
}

//forage: Main ACO loop. Performs the solution constructruction step, then updates distances, pheromones, probabilities.
void Colony::forage()
{ 
  //initialize variables and select start cities
  toVisit.assign(ARepeatCMap.begin(),ARepeatCMap.end());
  ACInt2.assign(ACKey.begin(),ACKey.end());
  thrust::fill(antVisits.begin(),
	       antVisits.end(),
	       0);
  thrust::sequence(tourMap.begin(),
		   tourMap.end(),
		   0,
		   numCities);
  thrust::transform(ARandom.begin(), 
		    ARandom.end(),
		    ARandom.begin(),
		    randStep());
  thrust::transform(ARandom.begin(),
		    ARandom.end(),
		    thrust::make_constant_iterator(numCities), 
		    thrust::make_permutation_iterator(antTours.begin(),tourMap.begin()), 
		    thrust::modulus<unsigned int>());
  thrust::transform(ACMapF.begin(),
		    ACMapF.end(),
		    thrust::make_permutation_iterator(antTours.begin(),tourMap.begin()),
		    AInt.begin(),
		    thrust::plus<int>());
  for(int x = 1; x < numCities; x++)
    {
      //update antVisits
      thrust::scatter(thrust::make_constant_iterator(x,0),
		      thrust::make_constant_iterator(x,numAnts), 
		      AInt.begin(),
		      antVisits.begin());
      thrust::remove_if(thrust::make_zip_iterator(thrust::make_tuple(toVisit.begin(),
								     ACInt2.begin())), 
			thrust::make_zip_iterator(thrust::make_tuple(toVisit.begin() + ((numCities-x + 1) * numAnts),
								     ACInt2.begin()+ ((numCities-x + 1) * numAnts))),
			antVisits.begin(),isX(x));
      //get probabilities
      thrust::transform(thrust::make_permutation_iterator(thrust::make_permutation_iterator(antTours.begin(),tourMap.begin()),ACInt2.begin()),
			thrust::make_permutation_iterator(thrust::make_permutation_iterator(antTours.end(),tourMap.end()),ACInt2.begin()+ ((numCities-x) * numAnts)),
			toVisit.begin(),
			ACInt.begin(),
			saxpy_functor(numCities));
      //update tour map
      thrust::transform(tourMap.begin(),
			tourMap.end(),
			tourMap.begin(),
			unaryPlus(1));
      //update random numbers
      thrust::transform(ACRandom.begin(),
			ACRandom.begin() + ((numCities-x + 1) * numAnts), 
			ACRandom.begin(),
			randStep());
      //select cities
      thrust::reduce_by_key(ACInt2.begin(),
			    ACInt2.begin()+ ((numCities-x) * numAnts),
			    thrust::make_zip_iterator(thrust::make_tuple(thrust::make_counting_iterator(0),
									 thrust::make_permutation_iterator(probabilities.begin(),ACInt.begin()),
									 ACRandom.begin())),
			    thrust::make_discard_iterator(),
			    thrust::make_zip_iterator(thrust::make_tuple(AInt.begin(),
									 AFloat.begin(),
									 AUnsignedInt.begin())),
			    thrust::equal_to<int>(),
			    treeSelect());
      thrust::gather(AInt.begin(),
		     AInt.end(),
		     toVisit.begin(),
		     thrust::make_permutation_iterator(antTours.begin(),tourMap.begin()));
    }
  computeAntDistances();
  updatePheromones();
  computeProbabilities();
}

//computeAntDistances: Computes the distances of each ant's tour, then updates records.
void Colony::computeAntDistances()
{
  //compute distances
  thrust::transform(antTours.begin(),
		    antTours.end(), 
		    thrust::make_permutation_iterator(antTours.begin(),distMap.begin()),
		    ACInt.begin(),
		    saxpy_functor(numCities));	
  thrust::gather(ACInt.begin(),
		 ACInt.end(),
		 distances.begin(),
		 ACFloat.begin());
  thrust::reduce_by_key(ACKey.begin(),
			ACKey.end(),
			ACFloat.begin(),
			thrust::make_discard_iterator(),
			antDistances.begin());
  //update bests
  int i = thrust::min_element(antDistances.begin(),
			      antDistances.end()) - antDistances.begin();
  thrust::gather(thrust::make_counting_iterator(i*numCities),
		 thrust::make_counting_iterator((i+1)*numCities),
		 antTours.begin(),
		 iterBestTour.begin());
  iterBestDist = antDistances[i];
  if(iterBestDist < globBestDist){
    reps = 0;
    globBestDist = iterBestDist;
    globBestTour.assign(iterBestTour.begin(),iterBestTour.end());
  }else{
    reps++;
  }
}

//greedyDistance: Returns the value of a simple greedy solution starting at city 0.
float Colony::greedyDistance()
{
  float distance;
  int i = 0;
  int init = i;
  thrust::device_vector<int> visits(numCities);
  thrust::fill(visits.begin(),
	       visits.end(),
	       1);
  thrust::device_vector<float> Cfloat(numCities);
  int j;
  for(int x = 1; x < numCities; x++){
    visits[i] = 0;
    thrust::transform(visits.begin(),
		      visits.end(),
		      thrust::make_permutation_iterator(distances.begin(),thrust::make_counting_iterator(i*numCities)),
		      Cfloat.begin(),
		      thrust::divides<float>());
    j = thrust::max_element(Cfloat.begin(),
			    Cfloat.end()) - Cfloat.begin();
    distance += distances[numCities * i + j];
    i = j;
  }
  distance += distances[numCities * i + init];
  return distance;
}

//computeProbabilities: Computes the probabilities from the distances and pheromones.
void Colony::computeProbabilities()
{
  thrust::transform(pheromones.begin(),
		    pheromones.end(),
		    distances.begin(),
		    probabilities.begin(),
		    prob_functor(beta));
}

void Colony::setBeta(float newBeta)
{
  beta = newBeta;
}

void Colony::setRho(float newRho)
{
  rho = newRho;
}

double Colony::getBeta()
{
  return beta;
}

double Colony::getRho()
{
  return rho;
}

int Colony::getNumAnts()
{
  return numAnts;
}

double Colony::getIterBestDist()
{
  return iterBestDist;
}

double Colony::getGlobBestDist()
{
  return globBestDist;
}

int Colony::getReps()
{
  return reps;
}

//Copyright (c) 2012, Peter Ahrens
//All rights reserved.
//
//Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
//
//    Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
//    Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
//    Neither the name of Excellants nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
//
//THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
