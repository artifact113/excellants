#include "hip/hip_runtime.h"
/****************************************
 * TSPReader.cu                         *
 * Dustin Tauxe and Peter Ahrens        *
 * Reads .tsp files                     *
 ****************************************/

#include "TSPReader.h"

//destructor
TSPReader::~TSPReader()
{	
  delete[] cityNames;
  delete[] Xcoords;
  delete[] Ycoords;
}

//read: Reads a given tsp file and extracts data.
bool TSPReader::read(char* filen)
{	
  ifstream infile(filen, ios_base::in);
  if(!infile){
    cout << "\n" << "Unable to open file: " << filen << "\n";
    return false;
  }
  string line;
  string tag;
  string value;
  while(infile.good()){
    getline(infile,line);
    if(line.length() > 1){
      if(!isprint(line[line.length()-1])) line.erase(line.length()-1,1);
      if(line.find(":") != string::npos){
	tag = line.substr(0,line.find(":"));
	value = line.substr(line.find(":") + 1, line.length() - line.find(":") - 1);
      }else{
	tag = line;
	value = "";
      }
      while(tag.find(" ") != string::npos){
	tag.replace(tag.find(" "),1,"");
      }
      while(value.find(" ") != string::npos){
	value.replace(value.find(" "),1,"");
      }
      if(tag == "NAME"){
	name = value;
      }else if(tag == "TYPE"){
	if(value != "TSP" && value != "STSP"){
	  cout << "\n" << "Invalid problem type: " << value << "\n";
	  return false;
	}
      }else if(tag == "DIMENSION"){
	numCities = atoi(value.c_str());
      }else if(tag == "EDGE_WEIGHT_TYPE"){
	if(value != "EUC_2D"){
	  cout << "\n" << "Invalid edge weight type: " << value << "\n";
	  return false;
	}
      }else if(tag == "NODE_COORD_SECTION"){
	//Set coord arrays to appropriate lengths
	cityNames = new string [numCities];
	Xcoords = new float [numCities];
	Ycoords = new float [numCities];
	for(int i = 0; infile.good() && i < numCities; i++){
	  getline(infile,line);
	  if(!isprint(line[line.length()-1])) line.erase(line.length()-1,1);
	  if(line == "EOF"){
	    return false;
	  }
	  cityNames[i] = line.substr(0,line.find(" "));
	  Xcoords[i] = atof(line.substr(line.find(" ") + 1, line.find_last_of(" ") - line.find(" ") - 1).c_str());
	  Ycoords[i] = atof(line.substr(line.find_last_of(" ") + 1, line.length() - line.find_last_of(" ") - 1).c_str());
	}
      }
						 
    }
  }
  calculateDistances();
  return true;
}

//calculateDistances: Calculates distances on the CPU.
void TSPReader::calculateDistances(){
  distances = thrust::host_vector<float> (numCities*numCities);
  float k;
  for(int i = 0; i < numCities; i++){
    for(int j = 0; j < numCities; j++){
      k = sqrt(pow(Xcoords[i]-Xcoords[j],2)+pow(Ycoords[i]-Ycoords[j],2));
      if(i == j)
	k = std::numeric_limits<float>::max();
      distances[i * numCities + j] = k;
    }
  }
}
	
string TSPReader::getName()
{ 
  return name; 
}
	
float* TSPReader::getXcoords()
{ 
  return Xcoords; 
}
	
float* TSPReader::getYcoords()
{ 
  return Ycoords; 
}
	
int TSPReader::getNumNodes()
{ 
  return numCities;
}
	
thrust::host_vector<float> TSPReader::getDistances()
{ 
  return distances; 
}

//Copyright (c) 2012, Peter Ahrens
//All rights reserved.
//
//Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
//
//    Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
//    Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
//    Neither the name of Excellants nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
//
//THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
